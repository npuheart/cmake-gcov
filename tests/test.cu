#include "hip/hip_runtime.h"
/// @date 2024-08-03
/// @file main.cu
/// @author Ma Pengfei (code@pengfeima.cn)
/// @version 0.1
/// @copyright Copyright (c) 2024 Ma Pengfei
///
/// @brief
///
///

#include <MnSystem/Cuda/HostUtils.hpp>
#include <cstdio>
#include <hip/hip_runtime.h>

using namespace mn;

__host__ __device__ void say_hello() { printf("Hello, world!\n"); }

__global__ void kernel() { say_hello(); }

void print_devices() {
    int         deviceCount = 0;
    hipError_t status      = hipGetDeviceCount(&deviceCount);

    if (status != hipSuccess) {
        std::cerr << "Error: hipGetDeviceCount failed with error code " << status << std::endl;
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        status = hipGetDeviceProperties(&deviceProp, device);

        if (status != hipSuccess) {
            std::cerr << "Error: hipGetDeviceProperties failed for device " << device << " with error code " << status
                      << std::endl;
            continue;
        }

        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total global memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Max threads per multiprocessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Warp size: " << deviceProp.warpSize << std::endl;
        std::cout << "  Max grid size: [" << deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1] << ", "
                  << deviceProp.maxGridSize[2] << "]" << std::endl;
        std::cout << "  Max block size: [" << deviceProp.maxThreadsDim[0] << ", " << deviceProp.maxThreadsDim[1] << ", "
                  << deviceProp.maxThreadsDim[2] << "]" << std::endl;
        std::cout << std::endl;
    }
}
int main_cuda() {
    // mn::Cuda::startup();
    kernel<<<2, 2>>>();
    hipDeviceSynchronize();
    check_cuda_errors(hipSetDevice(1));
    get_last_cuda_error("上一个错误");
    say_hello();
    print_devices();
    return 0;
}