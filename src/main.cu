/// @date 2024-08-02
/// @file main.cpp
/// @author Ma Pengfei (code@pengfeima.cn)
/// @version 0.1
/// @copyright Copyright (c) 2024 Ma Pengfei
///
/// @brief main runner
///
///


#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>

__host__ __device__ void say_hello() {
    printf("Hello, world!\n");
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<2, 2>>>();
    hipDeviceSynchronize();
    say_hello();

    // 检查 kernel 启动的错误
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // 检查 kernel 执行的错误
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA error during synchronization: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    return 0;
}
