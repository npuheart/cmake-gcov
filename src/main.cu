#include "hip/hip_runtime.h"
/// @date 2024-08-03
/// @file main.cu
/// @author Ma Pengfei (code@pengfeima.cn)
/// @version 0.1
/// @copyright Copyright (c) 2024 Ma Pengfei
/// 
/// @brief 
/// 
///

#include <cstdio>
#include <hip/hip_runtime.h>
#include <MnSystem/Cuda/HostUtils.hpp>

using namespace mn;

__host__ __device__ void say_hello() {
    printf("Hello, world!\n");
}

__global__ void kernel() {
    say_hello();
}

int main_cuda() {
    kernel<<<2, 2>>>();
    hipDeviceSynchronize();
    check_cuda_errors(hipSetDevice(1));
    get_last_cuda_error("上一个错误");
    say_hello();
    return 0;
}